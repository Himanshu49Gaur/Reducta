#include <stdio.h>
#include <stdlib.h>
#include "support.cu"
#include "kernel.cu"

int main(int argc, char* argv[])
{
    Timer timer;  // Timer for measuring execution time

    // Initialize host variables ----------------------------------------------
    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *in_h, *out_h;  // Host input and output arrays
    float *in_d, *out_d;  // Device input and output arrays
    unsigned in_elements, out_elements;  // Number of elements in input and output arrays
    hipError_t cuda_ret;  // CUDA return status variable
    dim3 dim_grid, dim_block;  // CUDA grid and block dimensions
    int i;  // Loop variable

    // Allocate and initialize host memory
    if(argc == 1) {
        in_elements = 1000000;  // Default input size if no argument is provided
    } else if(argc == 2) {
        in_elements = atoi(argv[1]);  // Convert argument to integer
    } else {
        // Invalid input handling
        printf("\n    Invalid input parameters!"
               "\n    Usage: ./reduction          # Input of size 1,000,000 is used"
               "\n    Usage: ./reduction <m>      # Input of size m is used"
               "\n");
        exit(0);
    }

    initVector(&in_h, in_elements);  // Initialize input vector with random values

    // Calculate the number of output elements (number of blocks needed for reduction)
    out_elements = in_elements / (BLOCK_SIZE << 1);
    if (in_elements % (BLOCK_SIZE << 1)) out_elements++;

    // Allocate host output memory
    out_h = (float*)malloc(out_elements * sizeof(float));
    if (out_h == NULL) FATAL("Unable to allocate host memory");

    stopTime(&timer); 
    printf("%f s\n", elapsedTime(timer));
    printf("    Input size = %u\n", in_elements);

    // Allocate device variables ----------------------------------------------
    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    // Allocate device memory for input array
    cuda_ret = hipMalloc((void**)&in_d, in_elements * sizeof(float));
    if (cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // Allocate device memory for output array
    cuda_ret = hipMalloc((void**)&out_d, out_elements * sizeof(float));
    if (cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    hipDeviceSynchronize();  // Synchronize device
    stopTime(&timer); 
    printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    // Copy input data from host to device
    cuda_ret = hipMemcpy(in_d, in_h, in_elements * sizeof(float), hipMemcpyHostToDevice);
    if (cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

    // Initialize device output array to zero
    cuda_ret = hipMemset(out_d, 0, out_elements * sizeof(float));
    if (cuda_ret != hipSuccess) FATAL("Unable to set device memory");

    hipDeviceSynchronize();  // Synchronize device
    stopTime(&timer); 
    printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    // Set CUDA kernel execution configuration
    dim_block.x = BLOCK_SIZE; dim_block.y = dim_block.z = 1;
    dim_grid.x  = out_elements; dim_grid.y = dim_grid.z = 1;

    // Launch the reduction kernel
    reduction<<<dim_grid, dim_block>>>(out_d, in_d, in_elements);

    // Ensure the kernel execution completes successfully
    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess)
        FATAL("Unable to launch/execute kernel");

    stopTime(&timer);
    printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    // Copy partial reduction results from device to host
    cuda_ret = hipMemcpy(out_h, out_d, out_elements * sizeof(float), hipMemcpyDeviceToHost);
    if (cuda_ret != hipSuccess)
        FATAL("Unable to copy memory to host");

    hipDeviceSynchronize();  // Synchronize device
    stopTime(&timer); 
    printf("%f s\n", elapsedTime(timer));

    // Debug: Print Partial Sums ----------------------------------------------
    printf("\nPartial sums from GPU:\n");
    for (i = 0; i < out_elements; i++) {
        printf("%f ", out_h[i]);  // DEBUG OUTPUT: Print partial sums from each block
    }
    printf("\n");

    // Final CPU accumulation -------------------------------------------------
    float final_sum = 0.0f;
    for (i = 0; i < out_elements; i++) {
        final_sum += out_h[i];  // Sum all partial sums to get the final result
    }

    printf("Final CPU sum after reduction: %f\n", final_sum);  // DEBUG OUTPUT

    // Verify correctness -----------------------------------------------------
    printf("Verifying results...\n");
    verify(in_h, in_elements, final_sum);  // Verify the computed sum against the expected value

    // Free memory ------------------------------------------------------------
    hipFree(in_d);  // Free device input memory
    hipFree(out_d);  // Free device output memory
    free(in_h);  // Free host input memory
    free(out_h);  // Free host output memory

    return 0;
}
// End of main.cu